#include "hip/hip_runtime.h"
#include <iostream>
#include "cudakmeans.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <cmath>

__device__ double distance(double *d_inputs, double *d_centroids, int idx1, int idx2, int dims)
{
	double dist = 0;

    for(int i = 0; i<dims; i++){
        dist += (d_centroids[idx2*dims + i] - d_inputs[idx1*dims + i] ) * (d_centroids[idx2*dims + i] - d_inputs[idx1*dims + i] );
    }
    
    dist = sqrt(dist);
    return dist;
}


__global__ void clusterAssignment(int *d_outputs, double *d_inputs, double *d_centroids,
								  int k, int n_vals, int dims, double *holdCentroid, double *numberCluster){

     int tid = blockDim.x * blockIdx.x + threadIdx.x;

     if (tid >= n_vals) return;
       
     double min_dist = DBL_MAX;
     int centroid_label = 0;
      
     double dist = 0;
       
        for (int i = 0; i < k; i++){
           dist = distance(d_inputs, d_centroids, tid, i, dims);
           
           if (dist < min_dist){
               min_dist = dist;
               centroid_label = i;     
           } 
        }


       __syncthreads();
       d_outputs[tid] = centroid_label;
	   
       for (int d = 0; d < dims; d++){      
            atomicAdd(&holdCentroid[centroid_label*dims + d],  d_inputs[tid*dims + d]);
       }
   
        atomicAdd(&numberCluster[centroid_label], 1.0);
}



__global__ void avgCentroids(double *d_centroids, int k, int dims, double *holdCentroid, double *numberCluster){

  
     int tid = blockDim.x * blockIdx.x + threadIdx.x;

	 if (tid >= k*dims) return;
	 
	 int clust = tid/dims;
	 d_centroids[tid] = holdCentroid[tid]/numberCluster[clust];   

	 
}



__global__ void warm_up_gpu(){
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;
  ib += ia + tid; 
}


bool converged(double* oldCentroids, double* currCentroids, int kclusters, int dims, double thres){
    
    
    double dist = 0;
    for (int k = 0; k< kclusters; k++){
        double temp_dist = 0;
        for(int i = 0; i<dims; i++){
            temp_dist = temp_dist + ( (currCentroids[k*dims + i] - oldCentroids[k*dims + i]  ) * (currCentroids[k*dims + i] - oldCentroids[k*dims + i] ));
        }

        temp_dist = sqrt(temp_dist);

        if (dist < temp_dist){
            dist = temp_dist;
        }

       }

     return (bool)(dist < (thres*thres));
    
    
}




void kernelKmeans(int* outputs, double* inputs, double* centroids, int k, int n_vals, int dims, int iters, double threshold){
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
	
    int *d_outputs;
    double *d_inputs;
    double *d_centroids;
    double *hold_centroids;
    double *numclusters;
    
    
    
     double prev_centroids[k*dims];
    
    
    
	hipFree(0); //used to warmup the gpu -- 
    hipMalloc((void**)&d_outputs , n_vals * sizeof(int));
    hipMalloc((void**)&d_inputs , n_vals * dims * sizeof(double));
    hipMalloc((void**)&d_centroids , k * dims * sizeof(double));
    hipMalloc((void**)&hold_centroids , k * dims * sizeof(double));
    hipMalloc((void**)&numclusters , k * sizeof(double));
  
    hipMemcpy(d_outputs, outputs,      n_vals * sizeof(int),         hipMemcpyHostToDevice);
    hipMemcpy(d_inputs, inputs,        n_vals * dims * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, centroids,  k * dims * sizeof(double),    hipMemcpyHostToDevice);
    
	

	const int T = 1024;
	const int grid_size = (n_vals + T) / T;
	warm_up_gpu<<<grid_size, T>>>();
	

   
    bool done = false;
    int iter = 0;
    while(!done){
         
         hipMemcpy(prev_centroids, d_centroids, k * dims * sizeof(double), hipMemcpyDeviceToHost);
         
         
         hipMemset(hold_centroids, 0,  k * dims * sizeof(double));
         hipMemset(numclusters, 0,  k * sizeof(double));
    
	
         clusterAssignment<<<grid_size,T>>>(d_outputs, d_inputs, d_centroids, k, n_vals, dims, hold_centroids, numclusters);
         hipDeviceSynchronize();
          
         avgCentroids<<<1, (k*dims)>>>(d_centroids, k, dims, hold_centroids, numclusters);
         hipDeviceSynchronize();
         
         
         hipMemcpy(centroids, d_centroids, k * dims * sizeof(double), hipMemcpyDeviceToHost);
        
      iter++;   
      done = (iter >= iters) || converged(prev_centroids, centroids, k, dims, threshold);
      }
    
    
    
    iters = iter;

       
    hipMemcpy(centroids, d_centroids, k * dims * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(outputs, d_outputs,     n_vals * sizeof(int),   hipMemcpyDeviceToHost);

    hipFree(hold_centroids);
    hipFree(numclusters);
    hipFree(d_outputs);
    hipFree(d_inputs);
    hipFree(d_centroids);
    
    
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    
    printf("%d,%lf\n", iters, time/iter);
       


   
}

